#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <stdio.h>
#include <stdlib.h>

#include "type.h"

#define TILE_WIDTH 32
__global__ void denseMatrixTranspose(float *A, float *A_T, int numARows, int numAColumns) {
	// The block and thread indices
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Identify the row and column of the P element
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;

	if (Row < numARows && Col < numAColumns) {
		A_T[Col*numARows + Row] = A[Row*numAColumns + Col];
	}
}

extern "C" void denseMatrixTranspose_wrapper(float **A, float **A_T, int numARows, int numAColumns) {
	float *d_A, *d_A_T;
	int i;

	/* Allocate device memory and copy host memory to device memory */
	hipMalloc((void **)&d_A, numARows*numAColumns * sizeof(float));
	hipMalloc((void **)&d_A_T, numARows*numAColumns * sizeof(float));

	/* Copy from host to device */
	for (i = 0; i < numARows; i++) {
		hipMemcpy(d_A + (i*numAColumns), A[i], numAColumns * sizeof(float), hipMemcpyHostToDevice);
	}

	/* Initialize the grid and block dimensions */
	dim3 DimGrid(ceil(1.0*numAColumns / TILE_WIDTH), ceil(1.0*numARows / TILE_WIDTH), 1);
	dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

	/* Launch the GPU kernel */
	denseMatrixTranspose<<< DimGrid, DimBlock >>>(d_A, d_A_T, numARows, numAColumns);
	hipDeviceSynchronize();
	/* Copyt the GPU memory back to the CPU */
	for (i = 0; i < numAColumns; i++) {
		hipMemcpy(A_T[i], d_A_T + (i*numARows), numARows * sizeof(float), hipMemcpyDeviceToHost);
	}

	/* Free the GPU memory */
	hipFree(d_A);
	hipFree(d_A_T);
}

__global__ void denseMatrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns) {
	__shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
	__shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];

	// The block and thread indices
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Identify the row and column of the P element
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	float Pvalue = 0;

	// Loop over the M and N tiles required to compute the P element
	for (int m = 0; m < (numAColumns - 1) / TILE_WIDTH + 1; m++) {
		// Collaborative loading of M and N tiles into shared memory
		if (Row < numARows && m*TILE_WIDTH + tx < numAColumns) {
			subTileA[ty][tx] = A[Row*numAColumns + m*TILE_WIDTH + tx];
		}
		else {
			subTileA[ty][tx] = 0;
		}
		if (m*TILE_WIDTH + ty < numBRows && Col < numBColumns) {
			subTileB[ty][tx] = B[(m*TILE_WIDTH + ty)*numBColumns + Col];
		}
		else {
			subTileB[ty][tx] = 0;
		}

		__syncthreads();
		if (Row < numARows && Col < numBColumns) {
			for (int k = 0; k < TILE_WIDTH; k++) {
				Pvalue += subTileA[ty][k] * subTileB[k][tx];
			}
		}

		__syncthreads();
	}

	if (Row < numARows && Col < numBColumns)
		C[Row*numBColumns + Col] = Pvalue;
}

extern "C" void denseMatrixMultiply_wrapper(float **A, float **B, float **C, int numARows, int numAColumns, int numBRows, int numBColumns) {
	float *d_A, *d_B, *d_C;
	int i;

	/* Allocate device memory and copy host memory to device memory */
	hipMalloc((void **)&d_A, numARows*numAColumns * sizeof(float));
	hipMalloc((void **)&d_B, numBRows*numBColumns * sizeof(float));
	hipMalloc((void **)&d_C, numARows*numBColumns * sizeof(float));

	/* Copy from host to device */
	for (i = 0; i < numARows; i++) {
		hipMemcpy(d_A + (i*numAColumns), A[i], numAColumns * sizeof(float), hipMemcpyHostToDevice);
	}
	for (i = 0; i < numBRows; i++) {
		hipMemcpy(d_B + (i*numBColumns), B[i], numBColumns * sizeof(float), hipMemcpyHostToDevice);
	}

	/* Initialize the grid and block dimensions */
	dim3 DimGrid(ceil(1.0*numBColumns / TILE_WIDTH), ceil(1.0*numARows / TILE_WIDTH), 1);
	dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

	/* Launch the GPU kernel */
	denseMatrixMultiply<<< DimGrid, DimBlock >>>(d_A, d_B, d_C, numARows, numAColumns, numBRows, numBColumns);
	hipDeviceSynchronize();
	/* Copyt the GPU memory back to the CPU */
	for (i = 0; i < numARows; i++) {
		hipMemcpy(C[i], d_C + (i*numBColumns), numBColumns * sizeof(float), hipMemcpyDeviceToHost);
	}

	/* Free the GPU memory */
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

__global__ void sparseVector(const int num_rows,
	const int *ptr,
	const int * indices,
	const float * data,
	const float * x,
	float * y,
	int Div)
{

	//__shared__ float vals[num_rows * 32];

	//int tid = blockDim.x*blockIdx.x + threadIdx.x;
	//int warp_id = tid / 32;
	//int lane = tid & (32 - 1);

	//int row = warp_id;

	//if (row < num_rows) {
	//	int row_start = ptr[row];
	//	int row_end = ptr[row + 1];

	//	vals[threadIdx.x] = 0;
	//	for (int jj = row_start + lane; jj < row_end; jj += 32)
	//		vals[threadIdx.x] += data[jj] * x[indices[jj]];

	//	if (lane < 16) vals[threadIdx.x] += vals[threadIdx.x + 16];
	//	if (lane < 8) vals[threadIdx.x] += vals[threadIdx.x + 8];
	//	if (lane < 4) vals[threadIdx.x] += vals[threadIdx.x + 4];
	//	if (lane < 2) vals[threadIdx.x] += vals[threadIdx.x + 2];
	//	if (lane < 1) vals[threadIdx.x] += vals[threadIdx.x + 1];

	//	if (lane == 0) {
	//		y[row] += vals[threadIdx.x]/Div;
	//	}
	//}


	   //printf("%f\n", data[0]);
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(row < num_rows){
	    float dot = 0;
	    int row_start = ptr[row];
	    int row_end = ptr[row+1];
	
	    for (int jj = row_start; jj < row_end; jj++)
	    dot+= data[jj] * x[indices[jj]];
	
	    y[row] += dot / Div;
	   // printf("%f\n", y[row]);
	}
}

extern "C" void sparseVector_wrapper(sparse_rcs *A, float **B, float **C, int FR_RC, int Div) {
	float *d_v, *d_B, *d_C;
	int *d_j, *d_r;
	int i;
	/* Allocate device memory and copy host memory to device memory */
	hipMalloc((void **)&d_v, A->N * sizeof(float));
	hipMalloc((void **)&d_j, A->N * sizeof(int));
	hipMalloc((void **)&d_r, A->N * sizeof(int));
	hipMalloc((void **)&d_B, (FR_RC*FR_RC) * sizeof(float));
	hipMalloc((void **)&d_C, (FR_RC*A->m) * sizeof(float));

	/* Copy from host to device */
	hipMemcpy(d_v, A->v, A->N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, A->r, A->N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_j, A->j, A->N * sizeof(int), hipMemcpyHostToDevice);
	for (i = 0; i < FR_RC; i++) {
		hipMemcpy(d_B + (i*FR_RC), B[i], FR_RC * sizeof(float), hipMemcpyHostToDevice);
	}

	/* Initialize the grid and block dimensions */
	dim3 DimGrid(ceil(1.0*FR_RC / TILE_WIDTH), 1, 1);
	dim3 DimBlock(TILE_WIDTH, 1, 1);

	/* Launch the GPU kernel */
	for (i = 0; i < FR_RC; i++) {
		sparseVector <<< DimGrid, DimBlock >>> (A->m, d_r, d_j, d_v, d_B + (i*FR_RC), d_C + (i*FR_RC), Div);
		hipDeviceSynchronize();
	}
	/* Copyt the GPU memory back to the CPU */
	for (i = 0; i < FR_RC; i++) {
		hipMemcpy(C[i], d_C + (i*FR_RC), FR_RC * sizeof(float), hipMemcpyDeviceToHost);
	}

	/* Free the GPU memory */
	hipFree(d_v);
	hipFree(d_r);
	hipFree(d_j);
	hipFree(d_B);
	hipFree(d_C);
}

__global__ void cooToFullRank(float *v, int *i, int *j, float *Out, int len, int numCol) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < len) {
		Out[i[idx] * numCol + j[idx]] = v[idx];
	}
}

extern "C" void cooToFullRank_wrapper(sparse_coo *A_coo, float **A_FR) {
	float *d_v, *d_out;
	int *d_i, *d_j;
	int i;
	/* Allocate device memory and copy host memory to device memory */
	hipMalloc((void **)&d_v, A_coo->N * sizeof(float));
	hipMalloc((void **)&d_i, A_coo->N * sizeof(int));
	hipMalloc((void **)&d_j, A_coo->N * sizeof(int));
	hipMalloc((void **)&d_out, (A_coo->m*A_coo->n) * sizeof(float));

	/* Copy from host to device */
	hipMemcpy(d_v, A_coo->v, A_coo->N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_i, A_coo->i, A_coo->N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_j, A_coo->j, A_coo->N * sizeof(int), hipMemcpyHostToDevice);

	/* Initialize the grid and block dimensions */
	dim3 DimGrid(ceil(1.0*A_coo->N / TILE_WIDTH), 1, 1);
	dim3 DimBlock(TILE_WIDTH, 1, 1);

	/* Launch the GPU kernel */
	cooToFullRank <<< DimGrid, DimBlock >>> (d_v, d_i, d_j, d_out, A_coo->N, A_coo->n);
	hipDeviceSynchronize();

	/* Copyt the GPU memory back to the CPU */
	for (i = 0; i < A_coo->m; i++) {
		hipMemcpy(A_FR[i], d_out + (i*A_coo->n), A_coo->n * sizeof(float), hipMemcpyDeviceToHost);
	}

	/* Free the GPU memory */
	hipFree(d_v);
	hipFree(d_i);
	hipFree(d_j);
	hipFree(d_out);
}